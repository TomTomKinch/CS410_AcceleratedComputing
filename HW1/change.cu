/*******************************************
* vect_add.cu
* By: Thomas Kinch
* 4/11/18
* A basic add vector program using CUDA.
*******************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#define n 512

__global__ void add(float *d_a, float *d_b, float *d_c){
	d_c[blockIdx.x] = d_a[blockIdx.x] + d_b[blockIdx.x];
}

int main(){
	float *h_a, *h_b, *h_c; //Host variables

	//Malloc memory for host variables
	h_a = (float*)malloc(n * sizeof(float));
	h_b = (float*)malloc(n * sizeof(float));
	h_c = (float*)malloc(n * sizeof(float));
	
	float *d_a, *d_b, *d_c; //Device
	int size = n * sizeof(float);

	//Malloc memory for device variables
	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);

	//Memcpy - copy host values to device
	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
	
	//Add the Vectors
	add<<<size, 1>>>(d_a, d_b, d_c);

	//Copy device result to the host
	hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);


	//Define host variables
	for(int i = 0; i < 10; i++){
		printf("h_c[%d] = %.1f\n", i, h_c[i]);
	}
	
	//Free memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	free(h_a);
	free(h_b);
	free(h_c);

	return 0;
}
