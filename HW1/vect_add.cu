#include "hip/hip_runtime.h"
/*******************************************
* vect_add.cu
* By: Thomas Kinch
* 4/11/18
* A basic add vector program using CUDA.
*******************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#define threads 512

__global__ void vecAdd(int *d_a, int *d_b, int *d_c){
	int i = threadIdx.x;
	if(i < threads){
		d_c[i] = d_a[i] + d_b[i];
	}
} 

int main(){
	int *h_a, *h_b, *h_c; //Host variables	
	int *d_a, *d_b, *d_c; //Device
	int size = threads * sizeof(int);

	//Malloc memory for host variables
	h_a = (int*)malloc(size);
	h_b = (int*)malloc(size);
	h_c = (int*)malloc(size);
	
	//Define Host variables
	for(int i = 0; i < threads; i++){
		h_a[i] = i;
		h_b[i] = i;
	}

	//Malloc memory for device variables
	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);

	//Memcpy - copy host values to device
	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
		
	//Add the Vectors
	vecAdd<<<1, threads>>>(d_a, d_b, d_c);
	hipDeviceSynchronize();

	//Copy device result to the host
	hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

	//Print host_c variables
	for(int i = 0; i < 20; i++){
		printf("h_c[%d] = %d\n", i, h_c[i]);
	}
	
	//Free memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	free(h_a);
	free(h_b);
	free(h_c);

	return 0;
}
