#include "hip/hip_runtime.h"
/********************************************************
* life_seq_cuda.cu
* Modifed BY Thomas Kinch
* Date: 5/14/18
*******************************************************/

/* Compile with `gcc life.c`.
 * When CUDA-fied, compile with `nvcc life.cu`
 */

#include <hip/hip_runtime.h>
#include <stdlib.h> // for rand
#include <string.h> // for memcpy
#include <stdio.h> // for printf
#include <time.h> // for nanosleep

#define WIDTH 60
#define HEIGHT 40
#define threads 256

void fill_board(int *board, int width, int height) {
    int i;
    for (i=0; i<width*height; i++)
        board[i] = rand() % 2;
}

void print_board(int *board) {
    int x, y;
    for (y=0; y<HEIGHT; y++) {
        for (x=0; x<WIDTH; x++) {
            char c = board[y * WIDTH + x] ? '#':' ';
            printf("%c", c);
        }
        printf("\n");
    }
    printf("-----\n");
}


__global__ void step(int *current, int *next, int width, int height) {
	 
	 //Offset
	 const int offsets[8][2] = {{-1, 1},{0, 1},{1, 1},
										 {-1, 0},       {1, 0},
										 {-1,-1},{0,-1},{1,-1}};


	 // coordinates of the cell we're currently evaluating
	 int x = blockIdx.x * width + threadIdx.x;
	 int y = blockIdx.y * height + threadIdx.y;
      
    // offset index, neighbor coordinates, alive neighbor count
    int i, nx, ny, num_neighbors;

    // write the next board state
    for (y=0; y<height; y++) {
        for (x=0; x<width; x++) {

            // count this cell's alive neighbors
            num_neighbors = 0;
            for (i=0; i<8; i++) {
                // To make the board torroidal, we use modular arithmetic to
                // wrap neighbor coordinates around to the other side of the
                // board if they fall off.
                nx = (x + offsets[i][0] + width) % width;
                ny = (y + offsets[i][1] + height) % height;
                if (current[ny * width + nx]) {
                    num_neighbors++;
                }
            }

            // apply the Game of Life rules to this cell
            next[y * width + x] = 0;
            if ((current[y * width + x] && num_neighbors==2) ||
                    num_neighbors==3) {
                next[y * width + x] = 1;
            }
        }
    }
}

int main(int argc, const char *argv[]) {
    // parse the width and height command line arguments, if provided
    int width, height, iters, out;
    if (argc < 3) {
        printf("usage: life iterations 1=print\n"); 
        exit(1);
    }
    iters = atoi(argv[1]);
    out = atoi(argv[2]);
    if (argc == 5) {
        width = atoi(argv[3]);
        height = atoi(argv[4]);
        printf("Running %d iterarions at %d by %d pixels.\n", iters, width, height);
    } else {
        width = WIDTH;
        height = HEIGHT;
    }

    struct timespec delay = {0, 125000000}; // 0.125 seconds
    struct timespec remaining;
    // The two boards 
    int *current, *next, many=0;
	 int *gpu_curr, *gpu_next;

    size_t board_size = sizeof(int) * width * height;
    current = (int *) malloc(board_size); // same as: int current[width * height];
    next = (int *) malloc(board_size);    // same as: int next[width *height];
 

    // Initialize the global "current".
    fill_board(current, width, height);
	 
	 /**************
	 * Cuda Malloc
	 **************/
	 hipMalloc((void**)&gpu_curr, board_size);
	 hipMalloc((void**)&gpu_next, board_size);

	 //Begin Iterations
	 clock_t begin, end;
	 double time_spent = 0;
    while (many<iters) {
        many++;
        if (out==1)
			   //Print Current Board
            print_board(current);
 
				/************
				* Cuda memcpy - copy cpu variables to gpu
				*************/
				hipMemcpy(gpu_curr, current, board_size, hipMemcpyHostToDevice);
				hipMemcpy(gpu_next, next, board_size, hipMemcpyHostToDevice);

				//Utilize Step On GPU
				begin = clock();
				step<<<1, threads>>>(gpu_curr, gpu_next, width, height);
				//hipDeviceSynchronize();
				end = clock();
				time_spent += (double)(end - begin) / CLOCKS_PER_SEC;
				hipDeviceSynchronize();
				//Copy GPU values back to CPU
				hipMemcpy(current, gpu_curr, board_size, hipMemcpyDeviceToHost);
				hipMemcpy(next, gpu_next, board_size, hipMemcpyDeviceToHost);
				
				memcpy(current, next, board_size);
/*
        //evaluate the `current` board, writing the next generation into `next`.
        step(current, next, width, height);
        // Copy the next state, that step() just wrote into, to current state
        memcpy(current, next, board_size);

        // copy the `next` to CPU and into `current` to be ready to repeat the process

*/
        // We sleep only because textual output is slow and the console needs
        // time to catch up. We don't sleep in the graphical X11 version.
        if (out==1)
            nanosleep(&delay, &remaining);
    }
	 printf("Runtime: %f Seconds\n", time_spent); 

	 //Free Cuda Memory
	 hipFree(gpu_curr);
	 hipFree(gpu_next);

    return 0;
}
