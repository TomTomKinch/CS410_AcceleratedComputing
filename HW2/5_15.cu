#include "hip/hip_runtime.h"
/****************
 * 5_15.cu
 *
 ***************/

 __shared__ float partialSum[SIZE]; 
 partialSum[threadIdx.x] = X[blockIdx.x * blockDim.x + threadIdx.x];
 unsigned int t = threadIdx.x;
 for (unsigned int stride = blockDim.x/2; stride >= 1; strdie = stride>>1)
 {
 
 __syncthreads();
 if (t < stride)
	partialSum(t) += partialSum(t+stride);
 }
