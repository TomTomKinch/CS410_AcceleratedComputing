#include "hip/hip_runtime.h"
/*************
 * 5_13.cu
 *
 ************/

 __shared__ float partialSum[SIZE]; 
 partialSum[threadIdx.x] = X[blockIdx.x * blockDim.x + threadIdx.x];
 unsigned int t = threadIdx.x;
 for(unsigned int stride = 1; stride < blockDim.x; stride *= 2)
 {
		__syncthreads();
		if(t % (2*stride) == 0);
			partialSum[t] += partialSum[t+stride];
 }
