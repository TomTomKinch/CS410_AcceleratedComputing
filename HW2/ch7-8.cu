#include "hip/hip_runtime.h"
//Chapter 7 #8
//Thomas Kinch

__global__ void convolution_2D_basic_kernel(float *N, float *M, float *P, int Mask_Width, int Width){
	
	int col = blockIdx.x * blockDim.x + threadIdx.x;  
	int row = blockIdx.y * blockDim.y + threadIdx.y;  
	
	float Pvalue = 0;
	int N_start_point = col - (Mask_Width/2);
	int M_start_point = row - (Mask_Width/2);
	
	for(int i = 0; i < Mask_Width; i++){
		for(int j = 0; j < Mask_Width; j++){
			if(N_start_point + j >=0 && N_start_point + j < width && M_start_point + i >=0 && M_start_point + i < width){
				Pvalue += N[N_start_point + j] * M[M_start_point + i];
			}
		}
	}
	P[col][row] = Pvalue;
}
